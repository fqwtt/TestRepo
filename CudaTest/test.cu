#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>

using namespace std;
// Kernel definition
class Person {
public:
	int age = 10;
};
__global__ void VecAdd(int* A, int* B, int* C)
{
	int i = threadIdx.x;
	C[i] = A[i] + B[i];
	printf("%d,%d\n", threadIdx.y, threadIdx.x);
}
__global__ void PersonAdd(Person* A, Person* B, Person* C)
{
	int i = threadIdx.x;
	C[i].age = A[i].age + B[i].age;
	printf("%d,%d\n", threadIdx.y, threadIdx.x);
}
int main()
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	int size = 3;


	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << endl;
		//exit(0);
	}
	Person A[3];
	Person B[3];
	Person C[3];
	hipMalloc((void**)&dev_c, size * sizeof(int));
	hipMalloc((void**)&dev_a, size * sizeof(int));
	hipMalloc((void**)&dev_b, size * sizeof(int));
	hipMemcpy(dev_a, A, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, size * sizeof(int), hipMemcpyHostToDevice);
	dim3 threadsPerBlock(3, 2);

	// N ���̵߳��ں˵���
	VecAdd << <1, threadsPerBlock >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(C, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; i++) {
		cout << C[i].age << endl;
	}
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
	return 0;
}