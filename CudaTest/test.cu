#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#define THREAD_PER_BLOCK 256

using namespace std;

class Person {
public:
	Person() = default;

	Person(int age, double height) {
		this->age = age;
		this->height = height;
	}
	int age = 10;
	double height = 9.9;
};
// Kernel definition
__global__ void PersonAddKernel(Person* A, Person* B, Person* C)
{
	int i = threadIdx.x;
	int j = blockIdx.x;
	int id = j * THREAD_PER_BLOCK + i;
	C[id].age = A[id].age + B[id].age;
	C[id].height = A[id].height + B[id].height;
}

int main()
{
	Person* dev_a = NULL;
	Person* dev_b = NULL;
	Person* dev_c = NULL;

	int block_size = 2;
	int thread_size = THREAD_PER_BLOCK;
	int size = block_size * thread_size;
	hipSetDevice(0);
	unique_ptr<Person[]> A = make_unique<Person[]>(size);
	unique_ptr<Person[]> B = make_unique<Person[]>(size);
	unique_ptr<Person[]> C = make_unique<Person[]>(size);


	hipMalloc((void**)&dev_c, size * sizeof(Person));
	hipMalloc((void**)&dev_a, size * sizeof(Person));
	hipMalloc((void**)&dev_b, size * sizeof(Person));
	hipMemcpy(dev_a, A.get(), size * sizeof(Person), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B.get(), size * sizeof(Person), hipMemcpyHostToDevice);

	//system("pause");
	// N ���̵߳��ں˵���
	PersonAddKernel << <block_size, thread_size >> > (dev_a, dev_b, dev_c);
	hipDeviceSynchronize();
	hipMemcpy(C.get(), dev_c, size * sizeof(Person), hipMemcpyDeviceToHost);
	for (int i = 0; i < size; i++) {
		cout << C[i].age << " " << C[i].height << endl;
	}
	
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipDeviceReset();
	return 0;
}